#include <iostream>
#include <vector>
#include <cassert>
#include <cstdlib>
#include <hip/hip_runtime.h>



using std::vector;
using std::cout;

__global__ void naiveMatrixMul(const float *a, const float *b, float *c, int N) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    float sum = 0.0f;
    if (row < N && col < N) {
        for (int k = 0; k < N; k++) {
            sum += a[row * N + k] * b[k * N + col];
        }
        c[row * N + col] = sum;
    }
}

int main() {
    int N = 4096; // Matrix size
    size_t bytes = N * N * sizeof(float);

    vector<float> h_a(N * N), h_b(N * N), h_c(N * N);
    float *d_a, *d_b, *d_c;

    hipMalloc(&d_a, bytes);
    hipMalloc(&d_b, bytes);
    hipMalloc(&d_c, bytes);

    hipMemcpy(d_a, h_a.data(), bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b.data(), bytes, hipMemcpyHostToDevice);

    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((N + threadsPerBlock.x - 1) / threadsPerBlock.x, (N + threadsPerBlock.y - 1) / threadsPerBlock.y);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    naiveMatrixMul<<<blocksPerGrid, threadsPerBlock>>>(d_a, d_b, d_c, N);

    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    cout << "Time elapsed: " << milliseconds << " ms\n";

    hipMemcpy(h_c.data(), d_c, bytes, hipMemcpyDeviceToHost);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}
