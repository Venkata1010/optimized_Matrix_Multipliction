#include <iostream>
#include <vector>
#include <cassert>
#include <cstdlib>
#include <hip/hip_runtime.h>



#define TILE_SIZE 32

using std::vector;
using std::cout;

__global__ void tiledMatrixMul(const float *a, const float *b, float *c, int N) {
    __shared__ float tile_a[TILE_SIZE][TILE_SIZE];
    __shared__ float tile_b[TILE_SIZE][TILE_SIZE];

    int tx = threadIdx.x, ty = threadIdx.y;
    int row = blockIdx.y * TILE_SIZE + ty;
    int col = blockIdx.x * TILE_SIZE + tx;

    float sum = 0.0f;
    for (int i = 0; i < (N + TILE_SIZE - 1) / TILE_SIZE; ++i) {
        if (i * TILE_SIZE + tx < N && row < N) 
            tile_a[ty][tx] = a[row * N + i * TILE_SIZE + tx];
        else
            tile_a[ty][tx] = 0.0;

        if (i * TILE_SIZE + ty < N && col < N) 
            tile_b[ty][tx] = b[(i * TILE_SIZE + ty) * N + col];
        else
            tile_b[ty][tx] = 0.0;

        __syncthreads();

        for (int k = 0; k < TILE_SIZE; ++k)
            sum += tile_a[ty][k] * tile_b[k][tx];

        __syncthreads();
    }
    if (row < N && col < N)
        c[row * N + col] = sum;
}

int main() {
    int N = 4096; // Matrix size
    size_t bytes = N * N * sizeof(float);

    vector<float> h_a(N * N), h_b(N * N), h_c(N * N);
    float *d_a, *d_b, *d_c;

    hipMalloc(&d_a, bytes);
    hipMalloc(&d_b, bytes);
    hipMalloc(&d_c, bytes);

    hipMemcpy(d_a, h_a.data(), bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b.data(), bytes, hipMemcpyHostToDevice);

    dim3 threadsPerBlock(TILE_SIZE, TILE_SIZE);
    dim3 blocksPerGrid((N + threadsPerBlock.x - 1) / threadsPerBlock.x, (N + threadsPerBlock.y - 1) / threadsPerBlock.y);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    tiledMatrixMul<<<blocksPerGrid, threadsPerBlock>>>(d_a, d_b, d_c, N);

    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    cout << "Time elapsed: " << milliseconds << " ms\n";

    hipMemcpy(h_c.data(), d_c, bytes, hipMemcpyDeviceToHost);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}
